#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <math.h>

#define D_FILENAME_ORIG "ibmfloatdata.bin"
#define D_FILENAME_DEST1 "restruct1floatdata.bin"
#define D_FILENAME_DEST2 "restruct2floatdata.bin"
#define D_FILENAME_DEST3 "restruct3floatdata.bin"

#include"./segUtils/segUtils.h"

#define D_MAX_MEMORY_BLOCK 1024*1024



// Restruturacao

__device__ void ibm_to_restruct_1_conv(unsigned char ibm[4], unsigned char to[4])
{
    unsigned char addSign = (to[0] && 0x01) >> 7;
    to[0] = ibm[0] >> 1 + addSign;
    to[1] = ibm[3];
    to[2] = ibm[1];
    to[3] = ibm[2];
}

__device__ void ibm_to_restruct_2_conv(char ibm[4], char to[4])
{
    to[0] = ibm[0];
    to[1] = ibm[3];
    to[2] = ibm[1];
    to[3] = ibm[2];
}

__device__ void ibm_to_restruct_3_conv(char ibm[4], char to[4])
{
    to[0] = ibm[0];
    to[1] = ibm[2];
    to[2] = ibm[3];
    to[3] = ibm[1];
}

__device__ void ibm_to_restruct(float *from, float* to, char type)
{
    //register int fconv, fmant, i, t;
    int i;

    for (i = 0; i < D_MAX_MEMORY_BLOCK; ++i)
    {
        switch (type)
        {
        case D_DATA_RESTRUCT_TP1:
            ibm_to_restruct_1_conv((unsigned char*)&from[i], (unsigned char*)&to[i]);
            break;

        case D_DATA_RESTRUCT_TP2:
            ibm_to_restruct_2_conv((char*)&from[i], (char*)&to[i]);
            break;

        case D_DATA_RESTRUCT_TP3:
            ibm_to_restruct_3_conv((char*)&from[i], (char*)&to[i]);
            break;

        default:
            break;
        }

    }
}


__global__ void restructKernel(float* data, float* newData1, float* newData2, float* newData3, long int blockFileSize)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < blockFileSize)
    {
        ibm_to_restruct(data, newData1, D_DATA_RESTRUCT_TP1);
        ibm_to_restruct(data, newData2, D_DATA_RESTRUCT_TP2);
        ibm_to_restruct(data, newData3, D_DATA_RESTRUCT_TP3);
    }
}

/*
void mostraVetores(float* c, int tamanho)
{
    for (int i = 0; i < tamanho; ++i)
    {
        printf("%d) (c = %f)", i, c[i]);

        if (!((i + 1) % 3))
            printf("\n");
        else
            printf("\t");
    }

    printf("\n");
}
*/


long int calculaBlockFileSize( long int fileSize, long int filePos)
{
    long int ret = fileSize-filePos;

    if (ret > D_MAX_MEMORY_BLOCK)
        ret = D_MAX_MEMORY_BLOCK;

    return ret;
}

int main()
{
    FILE* fpOrig, * fpDest1, * fpDest2, * fpDest3;
    float* originalData = NULL;
    float* newData1 = NULL;
    float* newData2 = NULL;
    float* newData3 = NULL;

    float* dev_data = 0;
    float* dev_newData1 = 0;
    float* dev_newData2 = 0;
    float* dev_newData3 = 0;

    long numThreads = 1024;
    long numBlocks = 1;

    ST_CabData cabFileOrig;
    ST_CabData cabFileDest;

    fopen_s(&fpOrig, D_FILENAME_ORIG, "rb");

    if (fread((void*)&cabFileOrig, sizeof(ST_CabData), 1, fpOrig) != 1)
    {
        printf("Erro de leitura (erro no cabecalho)\n");
        return 1;
    }

    if (cabFileOrig.type != D_DATA_IBM)
    {
        printf("Erro de leitura (tipo diferente de arquivo)\n");
        return 1;
    }

    cabFileDest.size = cabFileOrig.size;
    long int blockFileSize = 0;
    long int blockFilePos = 0;

    fopen_s(&fpDest1, D_FILENAME_DEST1, "wb+");
    fopen_s(&fpDest2, D_FILENAME_DEST2, "wb+");
    fopen_s(&fpDest3, D_FILENAME_DEST3, "wb+");

    cabFileDest.type = D_DATA_RESTRUCT_TP1;
    if (fwrite(&cabFileDest, sizeof(ST_CabData), 1, fpDest1) != 1)
    {
        printf("Erro de gravacao (cabecalho)\n");
        return 1;
    }

    cabFileDest.type = D_DATA_RESTRUCT_TP2;
    if (fwrite(&cabFileDest, sizeof(ST_CabData), 1, fpDest2) != 1)
    {
        printf("Erro de gravacao (cabecalho)\n");
        return 1;
    }

    cabFileDest.type = D_DATA_RESTRUCT_TP3;
    if (fwrite(&cabFileDest, sizeof(ST_CabData), 1, fpDest3) != 1)
    {
        printf("Erro de gravacao (cabecalho)\n");
        return 1;
    }

    originalData = (float*)malloc(D_MAX_MEMORY_BLOCK * sizeof(float));
    newData1 = (float*)malloc(D_MAX_MEMORY_BLOCK * sizeof(float));
    newData2 = (float*)malloc(D_MAX_MEMORY_BLOCK * sizeof(float));
    newData3 = (float*)malloc(D_MAX_MEMORY_BLOCK * sizeof(float));

    while (!feof(fpOrig) && !ferror(fpOrig) && blockFilePos< cabFileOrig.size)
    {

        blockFileSize = calculaBlockFileSize(cabFileOrig.size, blockFilePos);

        if (fread(originalData, sizeof(float), blockFileSize, fpOrig) != blockFileSize)
        {
            printf("Erro de leitura (erro nos dados)\n");
            return 1;
        }

        hipDeviceReset();

        hipMalloc((void**)&dev_data, blockFileSize * sizeof(float));
        hipMalloc((void**)&dev_newData1, blockFileSize * sizeof(float));
        hipMalloc((void**)&dev_newData2, blockFileSize * sizeof(float));
        hipMalloc((void**)&dev_newData3, blockFileSize * sizeof(float));

        // Copy input vectors from host memory to GPU buffers.
        hipMemcpy(dev_data, originalData, blockFileSize * sizeof(float), hipMemcpyHostToDevice);

        if (blockFileSize <= 1024)
        {
            numThreads = blockFileSize;
            numBlocks = 1;
        }
        else 
        {
            numThreads = 1024;
            numBlocks = ((long int)(blockFileSize / 1024)) + ((blockFileSize % 1024) ? 1 : 0);
        }

        restructKernel << <numBlocks, numThreads >> > (dev_data, dev_newData1, dev_newData2, dev_newData3, blockFileSize);

        hipDeviceSynchronize();
        hipMemcpy(newData1, dev_newData1, blockFileSize * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(newData2, dev_newData2, blockFileSize * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(newData3, dev_newData3, blockFileSize * sizeof(float), hipMemcpyDeviceToHost);


        if (fwrite(newData1, sizeof(float), blockFileSize, fpDest1) != blockFileSize)
        {
            printf("Erro de gravacao (dados)\n");
            return 1;
        }
        if (fwrite(newData2, sizeof(float), blockFileSize, fpDest2) != blockFileSize)
        {
            printf("Erro de gravacao (dados)\n");
            return 1;
        }
        if (fwrite(newData3, sizeof(float), blockFileSize, fpDest3) != blockFileSize)
        {
            printf("Erro de gravacao (dados)\n");
            return 1;
        }

        blockFilePos += blockFileSize;
    }

    hipFree(dev_data);
    hipFree(dev_newData1);
    hipFree(dev_newData2);
    hipFree(dev_newData3);
    hipDeviceReset();

    free(originalData);
    free(newData1);
    free(newData2);
    free(newData3);

    fclose(fpOrig);
    fclose(fpDest1);
    fclose(fpDest2);
    fclose(fpDest3);

    return 0;
}
